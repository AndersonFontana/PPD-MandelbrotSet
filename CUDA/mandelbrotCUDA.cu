#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

using namespace std;

#define THREADSPERBLOCK 1024

// __global__ void mandel(char *d_vet, int MAX_N, int MAX_COL, int MAX_ROW) {
__global__ void mandel(char *d_vet, int MAX_ROW, int MAX_COL, int MAX_NUM) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < MAX_ROW * MAX_COL) {

		int c = i % MAX_COL;
		int r = i / MAX_COL;
		int n = 0;

		float x=0, y=0, tmp;
		while ((x*x) + (y*y) < 4 && ++n < MAX_NUM) {
			tmp = x*x - y*y + ((float)c * 2 / MAX_COL - 1.5);
			y   = x * y * 2 + ((float)r * 2 / MAX_ROW - 1);
			x   = tmp;
		}

		d_vet[i] = (n == MAX_NUM ? '#' : '.');
	}
}

int main(int argc, char **argv) {

	int MAX_ROW;
	int MAX_COL;
	int MAX_NUM;

	if (argc == 4) {
		MAX_ROW = atoi(argv[1]);
		MAX_COL = atoi(argv[2]);
		MAX_NUM = atoi(argv[3]);
	}
	else
		exit(1);

	int MAX_TAM = MAX_ROW * MAX_COL;

	size_t i, size = MAX_TAM * sizeof(char);
	int nBlocks = ( MAX_TAM + THREADSPERBLOCK - 1 ) / THREADSPERBLOCK;

	char *h_vet = (char *)malloc(size);
	char *d_vet = (char *)malloc(size);

	hipSetDevice(0);

	hipMalloc((void**)&d_vet, size);

	mandel<<<nBlocks, THREADSPERBLOCK>>>(d_vet, MAX_ROW, MAX_COL, MAX_NUM);

	hipMemcpy(h_vet, d_vet, size, hipMemcpyDeviceToHost);
	
	hipFree(d_vet);

	for(i = 0; i < MAX_TAM; ++i) {
		cout << h_vet[i];
		if (i % MAX_COL == MAX_COL-1)
			cout << endl;
	}

	free(h_vet);
}
